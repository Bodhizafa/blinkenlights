// -*- coding: utf-8 -*-
/*
Copyright Landon Meernik
This work is licensed under the Creative Commons Attribution-NonCommercial-ShareAlike 4.0 International License.
To view a copy of this license, visit http://creativecommons.org/licenses/by-nc-sa/4.0/
or send a letter to Creative Commons, PO Box 1866, Mountain View, CA 94042, USA.
*/

/*
ACHTUNG!
ALLES TURISTEN UND NONTEKNISCHEN LOOKENPEEPERS!
DAS KOMPUTERMASCHINE IST NICHT FUR DER GEFINGERPOKEN UND MITTENGRABEN!
ODERWISE IST EASY TO SCHNAPPEN DER SPRINGENWERK, BLOWENFUSEN UND POPPENCORKEN MIT SPITZENSPARKEN.
IST NICHT FUR GEWERKEN BEI DUMMKOPFEN.
DER RUBBERNECKEN SIGHTSEEREN KEEPEN DAS COTTONPICKEN HANDER IN DAS POCKETS MUSS.
ZO RELAXEN UND WATSCHEN DER BLINKENLICHTEN.
*/



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
__global__
void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1<<20;
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    add<<<1, 1>>>(N, x, y);
    hipDeviceSynchronize();
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;
    hipFree(x);
    hipFree(y);

    return 0;
}
